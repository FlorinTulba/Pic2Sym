#include "hip/hip_runtime.h"
/************************************************************************************************
 The application Pic2Sym approximates images by a
 grid of colored symbols with colored backgrounds.

 Copyrights from the libraries used by the program:
 - (c) 2016 Boost (www.boost.org)
		License: <http://www.boost.org/LICENSE_1_0.txt>
			or doc/licenses/Boost.lic
 - (c) 2015 OpenCV (www.opencv.org)
		License: <http://opencv.org/license.html>
            or doc/licenses/OpenCV.lic
 - (c) 2015 The FreeType Project (www.freetype.org)
		License: <http://git.savannah.gnu.org/cgit/freetype/freetype2.git/plain/docs/FTL.TXT>
	        or doc/licenses/FTL.txt
 - (c) 1997-2002 OpenMP Architecture Review Board (www.openmp.org)
   (c) Microsoft Corporation (Visual C++ implementation for OpenMP C/C++ Version 2.0 March 2002)
		See: <https://msdn.microsoft.com/en-us/library/8y6825x5(v=vs.140).aspx>
 - (c) 1995-2013 zlib software (Jean-loup Gailly and Mark Adler - see: www.zlib.net)
		License: <http://www.zlib.net/zlib_license.html>
            or doc/licenses/zlib.lic
 
 (c) 2016, 2017 Florin Tulba <florintulba@yahoo.com>

 This program is free software: you can use its results,
 redistribute it and/or modify it under the terms of the GNU
 Affero General Public License version 3 as published by the
 Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 GNU Affero General Public License for more details.

 You should have received a copy of the GNU Affero General Public License
 along with this program ('agpl-3.0.txt').
 If not, see <http://www.gnu.org/licenses/agpl-3.0.txt>.
 ***********************************************************************************************/

#include "blurCUDA.h"
#include "boxBlurCUDA.h"
#include "streamsManager.h"
#include "util.h"

#include <hip/hip_runtime.h>

using namespace std;

/// Computes the initial rolling sums for each row / column
__global__
void initRollSums(fp *dataDev, unsigned rowsOrCols, unsigned providedColsOrRows, unsigned maskRadius) {
	const unsigned rowOrCol = blockIdx.x * blockDim.x + threadIdx.x,
		idxStartRowOrCol = rowOrCol,
		idxEndRowOrCol = (providedColsOrRows - 1U) * rowsOrCols + rowOrCol;
	const fp dataEndRowOrCol = dataDev[idxEndRowOrCol];
	if(rowOrCol < rowsOrCols) {
		fp rollingSum = dataDev[idxStartRowOrCol] * fp(maskRadius + 1U); // init the sum with (radius+1)*1st value
		for(unsigned colOrRow = 1U, idx = idxStartRowOrCol + rowsOrCols; colOrRow <= maskRadius;
				++colOrRow, idx += rowsOrCols)
			rollingSum += (idx < idxEndRowOrCol ? dataDev[idx] : dataEndRowOrCol); // add following radius values

		dataDev[idxStartRowOrCol] = rollingSum;
	}
}

void computeInitialRollingSums(fp *firstColOrRow, size_t firstColOrRowDataSz,
							   fp *nextRelevantColsOrRows, size_t nextRelevantColsOrRowsDataSz,
							   fp *dataDev, unsigned rowsOrCols, unsigned providedColsOrRows,
							   unsigned maskRadius, hipStream_t streamId) {
	CHECK_OP(hipMemcpyAsync((void*)&dataDev[rowsOrCols],
		(void*)nextRelevantColsOrRows,
		nextRelevantColsOrRowsDataSz, hipMemcpyHostToDevice, streamId));

	// Launch kernel computing the rolling sum in dataDev[0 .. rowsOrCols]
	enum { ThreadsPerBlock = 64 };
	initRollSums<<<1 + (int(rowsOrCols)-1)/ThreadsPerBlock, ThreadsPerBlock, 0ULL, streamId>>>
		(dataDev, rowsOrCols, providedColsOrRows, maskRadius);
	CHECK_OP(hipGetLastError());

	CHECK_OP(hipMemcpyAsync((void*)firstColOrRow, (void*)dataDev,
		firstColOrRowDataSz, hipMemcpyDeviceToHost, streamId));
}

/// Performs the box blur to the columns
__global__
void boxBlurCols(const fp* const __restrict__ imgBuf, fp* const __restrict__ blurredCols,
				const unsigned rows, const unsigned cols,
				const unsigned* const __restrict__ maskWidthsDev, const unsigned iterations) {
	extern __shared__ fp ioData[]; // 2*'rows' rows and blockDim.x columns
	const unsigned col = blockDim.x * blockIdx.x + threadIdx.x,
				maxInIdx = rows * blockDim.x;
	fp *inData = ioData, *outData = inData + maxInIdx, *aux = nullptr;

	// Copy data (columns accessed by this block from imgBuf) to shared memory
	if(col < cols) {
		for(unsigned row = 0U, inIdx = threadIdx.x, imgIdx = col; row < rows;
				++row, inIdx += blockDim.x, imgIdx += cols)
			inData[inIdx] = imgBuf[imgIdx];
	}
	__syncthreads();

	if(col < cols) {
		for(unsigned iter = 0U; iter < iterations;
				++iter, aux = inData, inData = outData, outData = aux) {
			const unsigned maskWidth = maskWidthsDev[iter],
						maskRadius = maskWidth >> 1;
			const fp colTop = inData[threadIdx.x],
					colBottom = inData[threadIdx.x + (rows-1U) * blockDim.x];

			/* Perform the box filtering (on each column) with a box of width maskWidth replicating the borders */

			// Setup of the rolling sum
			fp rollingSum = colTop * (1.f + maskRadius);
			for(unsigned row = 1U, inIdx = threadIdx.x + blockDim.x; row <= maskRadius; ++row, inIdx += blockDim.x)
				rollingSum += (inIdx < maxInIdx ? inData[inIdx] : colBottom);

			// Traversal of the column using the rolling sum
			int tailIdx = int(threadIdx.x - (int)maskRadius * blockDim.x);
			unsigned frontIdx = threadIdx.x + (1U + maskRadius) * blockDim.x;
			for(unsigned row = 0U, outIdx = threadIdx.x; row < rows;
					++row, outIdx += blockDim.x, tailIdx += (int)blockDim.x, frontIdx += blockDim.x) {
				outData[outIdx] = rollingSum;
				// There is a final rescaling at the end of 'boxBlurRows' (division by the product of all mask widths)

				rollingSum +=
					(frontIdx < maxInIdx ? inData[frontIdx] : colBottom) -
					(tailIdx > 0 ? inData[tailIdx] : colTop);
			}
		}
	}

	// Copy data (outData was swapped with inData) from shared memory to blurredCols
	__syncthreads();
	if(col < cols) {
		for(unsigned row = 0U, inIdx = threadIdx.x, outIdx = col; row < rows;
				++row, inIdx += blockDim.x, outIdx += cols)
			blurredCols[outIdx] = inData[inIdx];
	}
}

/// Performs the box blur to the rows. Affected by the uncoalesced global memory accesses
__global__
void boxBlurRows(fp* const __restrict__ ioDataGlob, const unsigned rows, const unsigned cols,
				const unsigned* const __restrict__ maskWidthsDev,
				const unsigned iterations, const fp scaler) {
	extern __shared__ fp prevVals[]; // maskRadius rows of blockDim.x columns. Each column is a circular buffer
	const unsigned row = blockDim.x * blockIdx.x + threadIdx.x,
				idxStartRow = row * cols,
				idxEndRow = idxStartRow + cols - 1U;
	if(row < rows) {
		// Data changes after each iteration, so the texture mechanism isn't practical, as it expect constant data.
		// The alternative is transposing the data matrix, applying boxBlurColumns on it and then transposing it back
		for(unsigned iter = 0U; iter < iterations; ++iter) {
			const fp startRow = ioDataGlob[idxStartRow],
					endRow = ioDataGlob[idxEndRow];
			const unsigned maskWidth = maskWidthsDev[iter],
						maskRadius = maskWidth >> 1,
						min_maskRadius_cols = min(maskRadius, cols);

			/* Perform the box filtering (on each row) with a box of width maskWidth replicating the borders */

			// Setup of the rolling sum
			fp rollingSum = startRow * (1.f + maskRadius);
			for(unsigned col = 1U, idx = idxStartRow + 1U; col <= maskRadius; ++col, ++idx)
				rollingSum += (idx < idxEndRow ? ioDataGlob[idx] : endRow);

			// Compute columns 0 .. min_maskRadius_cols-1
			unsigned frontIdx = idxStartRow + (1U + maskRadius),
					outIdx = idxStartRow,
					col = 0U;
			for(; col < min_maskRadius_cols; ++col, ++outIdx, ++frontIdx) {
				prevVals[threadIdx.x + col * blockDim.x] = ioDataGlob[outIdx];
				ioDataGlob[outIdx] = rollingSum;
				// There is a final rescaling at the end of this kernel (division by the product of all mask widths)

				rollingSum += (frontIdx < idxEndRow ? ioDataGlob[frontIdx] : endRow) - startRow;
			}

			// Compute columns min_maskRadius_cols .. cols-1
			for(unsigned tailIdxPrevVals = threadIdx.x + blockDim.x * (col % maskRadius); col < cols;
					++outIdx, ++frontIdx,
					tailIdxPrevVals = threadIdx.x + blockDim.x * (++col % maskRadius)) {
				fp temp = ioDataGlob[outIdx];
				ioDataGlob[outIdx] = rollingSum;
				// There is a final rescaling at the end of this kernel (division by the product of all mask widths)

				rollingSum +=
					(frontIdx < idxEndRow ? ioDataGlob[frontIdx] : endRow) - prevVals[tailIdxPrevVals];
				prevVals[tailIdxPrevVals] = temp;
			}
		}
	}

	// Wait for the completion of the iterations performing the blur by rows before scaling the result
	__syncthreads();
	for(unsigned row = blockDim.x * blockIdx.x, limRow = min(rows, row + blockDim.x); row < limRow; ++row)
		for(unsigned col = threadIdx.x, rowStart = row * cols; col < cols; col += blockDim.x)
			ioDataGlob[rowStart + col] *= scaler;
}

/// Launches the 2 kernels (horizontal and vertical) and manipulates the input and the output
void boxBlur(const fp *imgBuff, fp *result, fp *toBlurDev, fp *blurredDev,
			 unsigned rows, unsigned cols, size_t buffSz,
			 unsigned *maskWidthsDev, unsigned iterations,
			 unsigned largestMaskRadius, fp scaler,
			 hipStream_t streamId) {
	static const size_t fpSz = sizeof(fp);

// 	CHECK_OP(hipHostRegister((void*)imgBuff, buffSz, hipHostRegisterDefault));
	CHECK_OP(hipMemcpyAsync((void*)toBlurDev, (void*)imgBuff, buffSz, hipMemcpyHostToDevice, streamId));
// 	CHECK_OP(hipHostUnregister((void*)imgBuff));

	boxBlurCols<<<(cols + BoxBlurCUDA::BlockDimCols() - 1) / BoxBlurCUDA::BlockDimCols(),
				BoxBlurCUDA::BlockDimCols(),
				2U * BoxBlurCUDA::BlockDimCols() * rows * fpSz, // dynamic shared memory for in+out rows x blockDim tables of floats
				streamId>>>
		(toBlurDev, blurredDev, rows, cols, maskWidthsDev, iterations);
	CHECK_OP(hipGetLastError());

	boxBlurRows<<<(rows + BoxBlurCUDA::BlockDimRows() - 1) / BoxBlurCUDA::BlockDimRows(),
				BoxBlurCUDA::BlockDimRows(),
				BoxBlurCUDA::BlockDimRows() * largestMaskRadius * fpSz, // dynamic shared memory for blockDim x largest_mask_radius tables of floats
				streamId>>>
		(blurredDev, rows, cols, maskWidthsDev, iterations, scaler);
	CHECK_OP(hipGetLastError());

// 	CHECK_OP(hipHostRegister((void*)result, buffSz, hipHostRegisterDefault));
	CHECK_OP(hipMemcpyAsync((void*)result, (void*)blurredDev, buffSz, hipMemcpyDeviceToHost, streamId));
// 	CHECK_OP(hipHostUnregister((void*)result));

	CHECK_OP(hipStreamSynchronize(streamId));
}
